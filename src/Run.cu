#include "hip/hip_runtime.h"
#include <Shared.hh>
#include <Number.h>
#include <One.h>
#include <Two.h>
#include <Allocator.h>

#define ARRAY_SIZE 8

__global__
void getData( Number** numbers, int* data, int N )
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if( idx < N )
    {
        data[idx] = numbers[idx]->getNumber( );
    }
}

int main( )
{
    /// Validation data
    int* dataHost = new int[ARRAY_SIZE];
    int* dataDevice;
    hipMalloc( &dataDevice, sizeof( int ) * ARRAY_SIZE );

    /// ONES
    // Allocate on the GPU to use the virtual function
    Allocator< One >* onesAllocator = new Allocator< One >( ARRAY_SIZE );
    Number** ones = ( Number** )( onesAllocator->getObjects( ));

    // Run the kernel for the ones
    getData <<< 1, ARRAY_SIZE >>> ( ones, dataDevice, ARRAY_SIZE );

    // Get the data back
    hipMemcpy( dataHost, dataDevice, sizeof( int ) * ARRAY_SIZE,
               hipMemcpyDeviceToHost );
    for( int i = 0; i < ARRAY_SIZE; i++ )
        std::cout << "ones: " << dataHost[i] << std::endl;

    /// TWOS
    Allocator< Two >* twosAllocator = new Allocator<Two>( ARRAY_SIZE );
    Number** twos = ( Number** )( twosAllocator->getObjects( ));

    // Run the kernel for the twos
    getData <<< 1, ARRAY_SIZE >>> ( twos, dataDevice, ARRAY_SIZE );

    // Get the data back
    hipMemcpy( dataHost, dataDevice, sizeof( int ) * ARRAY_SIZE,
               hipMemcpyDeviceToHost);
    for( int i = 0; i < ARRAY_SIZE; i++ )
        std::cout << "twos: " << dataHost[i] << std::endl;

    return EXIT_SUCCESS;
}

#include "hip/hip_runtime.h"
#include <Shared.hh>
#include <Number.h>
#include <One.h>
#include <Two.h>
#include <Allocator.h>

#define ARRAY_SIZE 8

__global__
void getData( Number** numbers, int* data, int N )
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if( idx < N )
    {
        data[idx] = numbers[idx]->getNumber( );
    }
}

int main( )
{
    /// Validation data
    int* dataHost = new int[ARRAY_SIZE];
    int* dataDevice;
    hipMalloc( &dataDevice, sizeof( int ) * ARRAY_SIZE );

    /// ONES
    // Allocate on the GPU to use the virtual function
    Allocator< One >* onesAllocator = new Allocator< One >( ARRAY_SIZE );
    Number** ones = ( Number** )( onesAllocator->getObjects( ));

    // Run the kernel for the ones
    getData <<< 1, ARRAY_SIZE >>> ( ones, dataDevice, ARRAY_SIZE );

    // Get the data back
    hipMemcpy( dataHost, dataDevice, sizeof( int ) * ARRAY_SIZE,
               hipMemcpyDeviceToHost );
    for( int i = 0; i < ARRAY_SIZE; i++ )
        std::cout << "ones: " << dataHost[i] << std::endl;

    /// TWOS
    Allocator< Two >* twosAllocator = new Allocator<Two>( ARRAY_SIZE );
    Number** twos = ( Number** )( twosAllocator->getObjects( ));

    // Run the kernel for the twos
    getData <<< 1, ARRAY_SIZE >>> ( twos, dataDevice, ARRAY_SIZE );

    // Get the data back
    hipMemcpy( dataHost, dataDevice, sizeof( int ) * ARRAY_SIZE,
               hipMemcpyDeviceToHost);
    for( int i = 0; i < ARRAY_SIZE; i++ )
        std::cout << "twos: " << dataHost[i] << std::endl;

    return EXIT_SUCCESS;
}
























////Two* twoHost;
//__device__ Two* twoDevice;


///// KERNELS ////////////////////////////////////////////////////////////////////
//__global__
//void __kernel__getNumber(Number* numberDevice, int* number)
//{
//    int idx = threadIdx.x+blockDim.x*blockIdx.x;
//    if(idx == 0) {
//        *number = numberDevice->getNumber();
//    }
//}

//__global__
//void __kernel__allocateTwo()
//{
//    int idx = threadIdx.x+blockDim.x*blockIdx.x;
//    if(idx == 0) {
//        twoDevice = new Two(400);
//    }
//}

///// WRAPPERS ///////////////////////////////////////////////////////////////////
//void getNumber(Number* numberDevice, int* number)
//{
//    __kernel__getNumber <<< 1, 1 >>> (numberDevice, number);
//}

///// MAIN ///////////////////////////////////////////////////////////////////////
//int main()
//{
//    // Allocate the array that will get the one
//    int* resultHost = new int();
//    int* resultDevice;
//    hipMalloc(&resultDevice, sizeof(int));

////    /// Allocate memory for the device data
////    checkCudaErrors(hipHostMalloc((void**) &twoDevice,
////                                   sizeof(Two*), hipHostMallocWriteCombined));


//    __kernel__allocateTwo<<< 1, 1>>>();
//    hipDeviceSynchronize();
//    hipError_t error = hipGetLastError();
//    if(error!=hipSuccess)
//    {
//        fprintf(stderr,"1ERROR: validClassKernel: %s\n", hipGetErrorString(error) );
//    }



////    float *ah;
////        hipMalloc((void **)&ah, sz);
////        hipMemcpyToSymbol(HIP_SYMBOL("a"), &ah, sizeof(float *), size_t(0),hipMemcpyHostToDevice);

//    /// Get the address of thge device variable
//    Two* address;
//    hipMalloc((void **)&address, sizeof(Two));
//    hipGetSymbolAddress((void **)&address, twoDevice);

//    // checkCudaErrors(hipGetSymbolAddress((void**)&address, twoDevice));

//    /// Transfer the data to the address on the device
//    // checkCudaErrors(hipMemcpy(address, twoDevice, sizeof(Two*),hipMemcpyDeviceToDevice));









////    Two* devicePtr;
////    hipGetSymbolAddress((void**)&devicePtr, twoDevice);

////    //  hipMemcpyFromSymbol(twoHost, HIP_SYMBOL(twoDevice), sizeof(Two*), 0, hipMemcpyDeviceToHost);
////    hipDeviceSynchronize();
////    error = hipGetLastError();
////    if(error!=hipSuccess)
////    {
////        fprintf(stderr,"2ERROR: validClassKernel: %s\n", hipGetErrorString(error) );
////    }


//    getNumber(address, resultDevice);
//    hipDeviceSynchronize();
//    error = hipGetLastError();
//    if(error!=hipSuccess)
//    {
//        fprintf(stderr,"3ERROR: validClassKernel: %s\n", hipGetErrorString(error) );
//    }

//    // Transfer the array to the host
//    hipMemcpy(resultHost, resultDevice, sizeof(int),hipMemcpyDeviceToHost);

//    std::cout << "Result: " << *resultHost << std::endl;

//    return EXIT_SUCCESS;
//}

//#include <stdio.h>
//#include <hip/hip_runtime.h>
//#include "hip/hip_runtime_api.h"


////int host_x[4] = {1, 2, 3, 4};
////__device__ int dev_x[4];

////__global__ void kernel(int *d_var) { d_var[threadIdx.x] += 10; }

////int main(void)
////{
////    /* Declarations */
////    int data_size = 4 * sizeof(int);
////    int *address;

////    /* Allocate memory for `dev_x` using `hipHostMalloc` */
////    checkCudaErrors(hipHostMalloc((void**) &dev_x, data_size, hipHostMallocWriteCombined));

////    /* Get the address of the `__device__` variable `dev_x` */
////    checkCudaErrors(hipGetSymbolAddress((void**)&address, dev_x));

////    /* Transfer data to `address` on the device */
////    checkCudaErrors(hipMemcpy(address, host_x, data_size,hipMemcpyHostToDevice));

////    /* Launch the kernel for `address` */
////    kernel<<<1,4>>>(address);

////    checkCudaErrors(hipDeviceSynchronize());
////    getLastCudaError("wtf!");

////    checkCudaErrors(hipMemcpyFromSymbol(host_x, HIP_SYMBOL(dev_x), data_size, 0, hipMemcpyDeviceToHost));

////    for (int i=0; i< 4; i++){
////        printf("%d\n", host_x[i]);
////    }
////    return 0;
////}

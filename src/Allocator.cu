#include "hip/hip_runtime.h"
#include <Allocator.h>

template < class T >
__global__
void __kernel__allocateT( T** objects, const int N )
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if( idx < N )
    {
        objects[idx] = new T( );
    }
}

template < class T >
Allocator< T >::Allocator( const u_int64_t N )
{
    hipMalloc(( void** ) &_objects, sizeof( T* ) * N );
    __kernel__allocateT <<< 1, N >>> (_objects, N );
}

template < class T >
T** Allocator< T >::getObjects( ) const
{
    return _objects;
}

template class Allocator< One >;
template class Allocator< Two >;

